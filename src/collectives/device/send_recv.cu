#include "hip/hip_runtime.h"
#include "send_recv.h"
#include "common.h"
#include "collectives.h"

__global__ void ncclSendKernel(struct CollectiveArgs* args) {
    int dst = 1;
    const int tid = threadIdx.x;
    struct CollectiveArgs* args;
    const int nthreads = args->nThreads-WARP_SIZE;
    const int bid = args->bid;
    struct ncclDevComm* comm = args->comm;
    struct ncclChannel* channel = comm->channels+blockIdx.x;
    struct ncclRing* ring = &channel->ring;
    const ssize_t size = args->N;
    const int nranks = comm->nRanks;
    const int stepSize = channel->buffSize / (sizeof(T)*NCCL_STEPS);
    const int chunkSize = stepSize * ALLGATHER_CHUNKSTEPS;
    const T * __restrict__ thisInput = (const T*)args->ThisInput;
    /***********************IMPORTANT**************************/
    offset = chunkOffset + dst * size;
    ncclPrimitives<UNROLL, ALLGATHER_CHUNKSTEPS/ALLGATHER_SLICESTEPS, ALLGATHER_SLICESTEPS, T, 1, 1, FUNC> prims();
    prims.directSend(thisInput, 0, size);
}
